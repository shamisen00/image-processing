
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void device_strcpy(char *dst, const char *src) {
    while (*dst++ = *src++);
}

__global__ void kernel(char *A) {
    device_strcpy(A, "Hello, World!");
}

int main() {
   char *d_hello;
   char hello[32];

   hipMalloc((void**)&d_hello, 32);

   kernel<<<1,1>>>(d_hello);

   hipMemcpy(hello, d_hello, 32, hipMemcpyDeviceToHost);

   hipFree(d_hello);

   puts(hello);
}